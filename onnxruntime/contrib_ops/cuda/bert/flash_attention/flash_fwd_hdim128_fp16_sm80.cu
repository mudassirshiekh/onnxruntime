
#include <hip/hip_runtime.h>
// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.
#if USE_FLASH_ATTENTION

#include "contrib_ops/cuda/bert/flash_attention/flash_fwd_launch_template.h"

template <>
void run_mha_fwd_<cutlass::half_t, 128>(Flash_fwd_params& params, cudaStream_t stream) {
  run_mha_fwd_hdim128<cutlass::half_t>(params, stream);
}

#endif
